#include <iostream>
#include <vector>
#include <string>
#include <stdexcept>
#include <cstdint>
#include <map>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>
#include <memory>
#include <numeric>

#include <hip/hip_runtime.h>

// =======================================================================
// 1. CUDA Error Checking & Utility
// =======================================================================

#define CUDA_CHECK(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        hipDeviceReset(); \
        exit(EXIT_FAILURE); \
    } \
}

// RAII wrapper for CUDA memory to ensure cudaFree is always called.
template<typename T>
class CudaBuffer {
private:
    T* d_ptr = nullptr;
    size_t count = 0;

public:
    CudaBuffer() = default;

    explicit CudaBuffer(size_t n) {
        if (n > 0) {
            CUDA_CHECK(hipMalloc(&d_ptr, n * sizeof(T)));
            count = n;
        }
    }

    ~CudaBuffer() {
        if (d_ptr) {
            hipFree(d_ptr);
        }
    }

    // Disable copy semantics
    CudaBuffer(const CudaBuffer&) = delete;
    CudaBuffer& operator=(const CudaBuffer&) = delete;

    // Enable move semantics
    CudaBuffer(CudaBuffer&& other) noexcept : d_ptr(other.d_ptr), count(other.count) {
        other.d_ptr = nullptr;
        other.count = 0;
    }

    CudaBuffer& operator=(CudaBuffer&& other) noexcept {
        if (this != &other) {
            if (d_ptr) hipFree(d_ptr);
            d_ptr = other.d_ptr;
            count = other.count;
            other.d_ptr = nullptr;
            other.count = 0;
        }
        return *this;
    }

    T* get() { return d_ptr; }
    const T* get() const { return d_ptr; }
    size_t size() const { return count; }

    void copy_to_device(const T* h_data, size_t n) {
        if (n > count) throw std::runtime_error("Copy size exceeds buffer capacity.");
        CUDA_CHECK(hipMemcpy(d_ptr, h_data, n * sizeof(T), hipMemcpyHostToDevice));
    }

    void copy_to_host(T* h_data, size_t n) const {
        if (n > count) throw std::runtime_error("Copy size exceeds buffer capacity.");
        CUDA_CHECK(hipMemcpy(h_data, d_ptr, n * sizeof(T), hipMemcpyDeviceToHost));
    }
    
    void memset(int value) {
        if (d_ptr) {
            CUDA_CHECK(hipMemset(d_ptr, value, count * sizeof(T)));
        }
    }
};


__device__ inline int32_t floorDiv(int32_t a, int32_t n) {
    int32_t r = a / n;
    if ((a % n != 0) && ((a < 0) != (n < 0))) {
        r--;
    }
    return r;
}

// =======================================================================
// 2. Core Data Structures & Enums (Host and Device)
// =======================================================================

// --- Generic Enums ---
enum class BlockRotation {
    NONE = 0, CLOCKWISE_90 = 1, CLOCKWISE_180 = 2, COUNTERCLOCKWISE_90 = 3
};

// --- Ruined Portal Specific Enums ---
enum class BlockMirror { NONE, FRONT_BACK };
enum class BiomeCategory { MOUNTAINS = 1, DESERT = 2, JUNGLE = 3 };
enum class PortalType {
    PORTAL_1, PORTAL_2, PORTAL_3, PORTAL_4, PORTAL_5, PORTAL_6, PORTAL_7, PORTAL_8, PORTAL_9, PORTAL_10,
    GIANT_PORTAL_1, GIANT_PORTAL_2, GIANT_PORTAL_3
};

// --- Shipwreck Specific Enums ---
enum class ShipwreckType {
    INVALID = -1,
    RIGHTSIDEUP_BACKHALF, RIGHTSIDEUP_BACKHALF_DEGRADED, RIGHTSIDEUP_FRONTHALF, RIGHTSIDEUP_FRONTHALF_DEGRADED,
    RIGHTSIDEUP_FULL, RIGHTSIDEUP_FULL_DEGRADED, SIDEWAYS_BACKHALF, SIDEWAYS_BACKHALF_DEGRADED,
    SIDEWAYS_FRONTHALF, SIDEWAYS_FRONTHALF_DEGRADED, SIDEWAYS_FULL, SIDEWAYS_FULL_DEGRADED,
    UPSIDEDOWN_BACKHALF, UPSIDEDOWN_BACKHALF_DEGRADED, UPSIDEDOWN_FRONTHALF, UPSIDEDOWN_FRONTHALF_DEGRADED,
    UPSIDEDOWN_FULL, UPSIDEDOWN_FULL_DEGRADED, WITH_MAST, WITH_MAST_DEGRADED,
};

// --- Village Specific Enums (NEW) ---
enum class VillageType { PLAINS, DESERT, SAVANNA, TAIGA, SNOWY };
enum class VillageStartPiece {
    PLAINS_FOUNTAIN_01, PLAINS_MEETING_POINT_1, PLAINS_MEETING_POINT_2, PLAINS_MEETING_POINT_3,
    DESERT_MEETING_POINT_1, DESERT_MEETING_POINT_2, DESERT_MEETING_POINT_3,
    SAVANNA_MEETING_POINT_1, SAVANNA_MEETING_POINT_2, SAVANNA_MEETING_POINT_3, SAVANNA_MEETING_POINT_4,
    TAIGA_MEETING_POINT_1, TAIGA_MEETING_POINT_2,
    SNOWY_MEETING_POINT_1, SNOWY_MEETING_POINT_2, SNOWY_MEETING_POINT_3,
    UNKNOWN_PIECE
};

// --- Unified Constraint Structures ---
// MODIFIED: Added VILLAGE
enum class ConstraintType { SHIPWRECK, RUINED_PORTAL, VILLAGE };

struct RuinedPortalConstraintData {
    BlockRotation rotation;
    BlockMirror mirror;
    PortalType type;
    BiomeCategory category;
};

struct ShipwreckConstraintData {
    BlockRotation rotation;
    ShipwreckType type;
    bool isBeached;
};

// NEW: Village constraint data
struct VillageConstraintData {
    BlockRotation rotation;
    VillageStartPiece piece;
    VillageType type;
    bool is_abandoned;
};

// MODIFIED: Added VillageConstraintData to the union
struct Constraint {
    ConstraintType type;
    int32_t chunkX;
    int32_t chunkZ;
    union {
        ShipwreckConstraintData shipwreck;
        RuinedPortalConstraintData portal;
        VillageConstraintData village;
    };
};

// =======================================================================
// 3. Minecraft LCG & Constants
// =======================================================================

__constant__ int64_t LCG_MULT = 25214903917LL;
__constant__ int64_t LCG_ADD = 11LL;
__constant__ int64_t XOR_MASK = 25214903917LL;
__constant__ int64_t MASK_48 = (1LL << 48) - 1;
__constant__ int64_t LCG_MULT_INV = 246154705703781LL;
__constant__ int64_t MULT_A = 341873128712LL;
__constant__ int64_t MULT_B = 132897987541LL;
__device__ const int64_t PILLAR_MULT = 1540035429LL;
__device__ const int64_t PILLAR_ADD = 239479465LL;

// Shipwreck Constants
__constant__ int32_t SHIPWRECK_SPACING = 24;
__constant__ int32_t SHIPWRECK_SEPARATION = 4;
__constant__ int64_t SHIPWRECK_SALT = 165745295;
__constant__ int32_t OCEAN_TYPE_COUNT = 20;
__constant__ int32_t BEACHED_TYPE_COUNT = 11;
__constant__ ShipwreckType d_STRUCTURE_LOCATION_OCEAN[20];
__constant__ ShipwreckType d_STRUCTURE_LOCATION_BEACHED[11];

// Ruined Portal Constants
__constant__ int32_t PORTAL_SPACING = 40;
__constant__ int32_t PORTAL_SEPARATION = 15;
__constant__ int64_t RUINED_PORTAL_SALT = 34222645LL;

// Village Constants (NEW)
__constant__ int32_t VILLAGE_SPACING = 34;
__constant__ int32_t VILLAGE_SEPARATION = 8;
__constant__ int64_t VILLAGE_SALT = 10387312LL;

// =======================================================================
// 4. Device-Side Logic
// =======================================================================

// --- Standalone LCG for GPU ---
struct StandaloneChunkRand {
private:
    int64_t seed;
public:
    __device__ void setSeed(int64_t s) {
        seed = (s ^ XOR_MASK) & MASK_48;
    }
    __device__ int32_t next(int32_t bits) {
        seed = (seed * LCG_MULT + LCG_ADD) & MASK_48;
        return (int32_t)((uint64_t)seed >> (48 - bits));
    }
    __device__ int32_t nextInt(int32_t bound) {
        if (bound <= 0) return 0;
        if ((bound & -bound) == bound) return (int32_t)((bound * (int64_t)next(31)) >> 31);
        int32_t bits, val;
        do { bits = next(31); val = bits % bound; } while (bits - val + (bound - 1) < 0);
        return val;
    }
    __device__ int64_t nextLong() {
        return ((int64_t)next(32) << 32) + next(32);
    }
    __device__ float nextFloat() {
        return next(24) / 16777216.0f; // (1 << 24)
    }
    __device__ void setRegionSeed(int64_t structureSeed, int32_t regionX, int32_t regionZ, int64_t salt) {
        int64_t s = (long long)regionX * MULT_A + (long long)regionZ * MULT_B + structureSeed + salt;
        setSeed(s);
    }
    __device__ void setCarverSeed(int64_t worldSeed, int32_t chunkX, int32_t chunkZ) {
        setSeed(worldSeed);
        long long a = nextLong();
        long long b = nextLong();
        setSeed((long long)chunkX * a ^ (long long)chunkZ * b ^ worldSeed);
    }
};

// --- Validation Functions (called by kernels) ---

// NEW: Village property calculation, ported and adapted.
__device__ void get_village_properties_device(
    VillageStartPiece& out_piece, BlockRotation& out_rot, bool& out_abandoned,
    VillageType type, StandaloneChunkRand& rand
) {
    // Rotation is determined by the first call to nextInt(4) which is next(2)
    out_rot = (BlockRotation)rand.next(2);

    int t;
    out_piece = VillageStartPiece::UNKNOWN_PIECE;
    out_abandoned = false;

    switch (type) {
        case VillageType::PLAINS:
            t = rand.nextInt(204);
            if      (t <  50) { out_piece = VillageStartPiece::PLAINS_FOUNTAIN_01;     }
            else if (t < 100) { out_piece = VillageStartPiece::PLAINS_MEETING_POINT_1; }
            else if (t < 150) { out_piece = VillageStartPiece::PLAINS_MEETING_POINT_2; }
            else if (t < 200) { out_piece = VillageStartPiece::PLAINS_MEETING_POINT_3; }
            else { // Abandoned (2% chance)
                out_abandoned = true;
                if      (t < 201) { out_piece = VillageStartPiece::PLAINS_FOUNTAIN_01; }
                else if (t < 202) { out_piece = VillageStartPiece::PLAINS_MEETING_POINT_1; }
                else if (t < 203) { out_piece = VillageStartPiece::PLAINS_MEETING_POINT_2; }
                else              { out_piece = VillageStartPiece::PLAINS_MEETING_POINT_3; }
            }
            break;
        case VillageType::DESERT:
            t = rand.nextInt(250);
            if      (t <  98) { out_piece = VillageStartPiece::DESERT_MEETING_POINT_1; }
            else if (t < 196) { out_piece = VillageStartPiece::DESERT_MEETING_POINT_2; }
            else if (t < 245) { out_piece = VillageStartPiece::DESERT_MEETING_POINT_3; }
            else { // Abandoned (2% chance)
                out_abandoned = true;
                if      (t < 247) { out_piece = VillageStartPiece::DESERT_MEETING_POINT_1; }
                else if (t < 249) { out_piece = VillageStartPiece::DESERT_MEETING_POINT_2; }
                else              { out_piece = VillageStartPiece::DESERT_MEETING_POINT_3; }
            }
            break;
        case VillageType::SAVANNA:
            t = rand.nextInt(459);
            if      (t < 100) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_1; }
            else if (t < 150) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_2; }
            else if (t < 300) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_3; }
            else if (t < 450) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_4; }
            else { // Abandoned (2% chance)
                out_abandoned = true;
                if      (t < 452) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_1; }
                else if (t < 453) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_2; }
                else if (t < 456) { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_3; }
                else              { out_piece = VillageStartPiece::SAVANNA_MEETING_POINT_4; }
            }
            break;
        case VillageType::TAIGA:
            t = rand.nextInt(100);
            if      (t <  49) { out_piece = VillageStartPiece::TAIGA_MEETING_POINT_1; }
            else if (t <  98) { out_piece = VillageStartPiece::TAIGA_MEETING_POINT_2; }
            else { // Abandoned (2% chance)
                out_abandoned = true;
                if (t < 99) { out_piece = VillageStartPiece::TAIGA_MEETING_POINT_1; }
                else        { out_piece = VillageStartPiece::TAIGA_MEETING_POINT_2; }
            }
            break;
        case VillageType::SNOWY:
            t = rand.nextInt(306);
            if      (t < 100) { out_piece = VillageStartPiece::SNOWY_MEETING_POINT_1; }
            else if (t < 150) { out_piece = VillageStartPiece::SNOWY_MEETING_POINT_2; }
            else if (t < 300) { out_piece = VillageStartPiece::SNOWY_MEETING_POINT_3; }
            else { // Abandoned (2% chance)
                out_abandoned = true;
                if      (t < 302) { out_piece = VillageStartPiece::SNOWY_MEETING_POINT_1; }
                else if (t < 303) { out_piece = VillageStartPiece::SNOWY_MEETING_POINT_2; }
                else              { out_piece = VillageStartPiece::SNOWY_MEETING_POINT_3; }
            }
            break;
    }
}

// NEW: Full validation function for a village constraint
__device__ bool check_village_full(int64_t structureSeed, const Constraint& constraint, StandaloneChunkRand& rand) {
    // 1. Check if the chunk coordinates are correct for the region
    int32_t regX = floorDiv(constraint.chunkX, VILLAGE_SPACING);
    int32_t regZ = floorDiv(constraint.chunkZ, VILLAGE_SPACING);
    rand.setRegionSeed(structureSeed, regX, regZ, VILLAGE_SALT);
    
    int32_t offset = VILLAGE_SPACING - VILLAGE_SEPARATION;
    if (regX * VILLAGE_SPACING + rand.nextInt(offset) != constraint.chunkX) return false;
    if (regZ * VILLAGE_SPACING + rand.nextInt(offset) != constraint.chunkZ) return false;

    // 2. Check the properties based on the chunk-specific seed
    rand.setCarverSeed(structureSeed, constraint.chunkX, constraint.chunkZ);

    VillageStartPiece found_piece;
    BlockRotation found_rot;
    bool found_abandoned;

    get_village_properties_device(found_piece, found_rot, found_abandoned, constraint.village.type, rand);

    return found_piece == constraint.village.piece &&
           found_rot == constraint.village.rotation &&
           found_abandoned == constraint.village.is_abandoned;
}

__device__ bool check_shipwreck_full(int64_t structureSeed, const Constraint& constraint, StandaloneChunkRand& rand) {
    int32_t regX = floorDiv(constraint.chunkX, SHIPWRECK_SPACING);
    int32_t regZ = floorDiv(constraint.chunkZ, SHIPWRECK_SPACING);
    rand.setRegionSeed(structureSeed, regX, regZ, SHIPWRECK_SALT);
    
    int32_t offset = SHIPWRECK_SPACING - SHIPWRECK_SEPARATION;
    if (regX * SHIPWRECK_SPACING + rand.nextInt(offset) != constraint.chunkX) return false;
    if (regZ * SHIPWRECK_SPACING + rand.nextInt(offset) != constraint.chunkZ) return false;

    rand.setCarverSeed(structureSeed, constraint.chunkX, constraint.chunkZ);
    if (static_cast<BlockRotation>(rand.nextInt(4)) != constraint.shipwreck.rotation) return false;
    
    ShipwreckType type;
    if (constraint.shipwreck.isBeached) {
        type = d_STRUCTURE_LOCATION_BEACHED[rand.nextInt(BEACHED_TYPE_COUNT)];
    } else {
        type = d_STRUCTURE_LOCATION_OCEAN[rand.nextInt(OCEAN_TYPE_COUNT)];
    }
    return type == constraint.shipwreck.type;
}

__device__ bool check_portal_full(int64_t structureSeed, const Constraint& constraint, StandaloneChunkRand& rand) {
    int32_t regX = floorDiv(constraint.chunkX, PORTAL_SPACING);
    int32_t regZ = floorDiv(constraint.chunkZ, PORTAL_SPACING);
    rand.setRegionSeed(structureSeed, regX, regZ, RUINED_PORTAL_SALT);

    int32_t offset = PORTAL_SPACING - PORTAL_SEPARATION;
    if (regX * PORTAL_SPACING + rand.nextInt(offset) != constraint.chunkX) return false;
    if (regZ * PORTAL_SPACING + rand.nextInt(offset) != constraint.chunkZ) return false;

    rand.setCarverSeed(structureSeed, constraint.chunkX, constraint.chunkZ);
    
    switch (constraint.portal.category) {
        case BiomeCategory::DESERT: break;
        case BiomeCategory::JUNGLE: rand.nextFloat(); break;
        case BiomeCategory::MOUNTAINS:
            if (rand.nextFloat() >= 0.5f) { rand.nextFloat(); }
            break;
    }

    if (rand.nextFloat() < 0.05f) { // Giant Portal
        if (static_cast<PortalType>(static_cast<int>(PortalType::GIANT_PORTAL_1) + rand.nextInt(3)) != constraint.portal.type) return false;
    } else { // Regular Portal
        if (static_cast<PortalType>(static_cast<int>(PortalType::PORTAL_1) + rand.nextInt(10)) != constraint.portal.type) return false;
    }

    if (static_cast<BlockRotation>(rand.nextInt(4)) != constraint.portal.rotation) return false;
    if (((rand.nextFloat() < 0.5f) ? BlockMirror::NONE : BlockMirror::FRONT_BACK) != constraint.portal.mirror) return false;
    
    return true;
}


// =======================================================================
// 5. Host-Side Abstraction Layer
// =======================================================================

// Base class for defining a searchable Minecraft structure.
class IStructure {
public:
    virtual ~IStructure() = default;
    virtual std::string get_name() const = 0;
    virtual ConstraintType get_type() const = 0;
    virtual bool try_parse_constraint(const std::vector<std::string>& parts, Constraint& out_constraint) const = 0;
    virtual void initialize_device_constants() const = 0;
    virtual bool has_fast_filter() const { return false; }
    virtual bool has_reversing_kernel() const { return false; }
};

// NEW: Concrete implementation for Villages.
class VillageStructure : public IStructure {
private:
    std::map<std::string, VillageStartPiece> name_to_piece;
    std::map<int, VillageType> biome_id_to_type;
public:
    VillageStructure() {
        // Map user biome IDs to internal enum
        biome_id_to_type[1] = VillageType::PLAINS;
        biome_id_to_type[2] = VillageType::SNOWY;
        biome_id_to_type[3] = VillageType::TAIGA;
        biome_id_to_type[4] = VillageType::SAVANNA;
        biome_id_to_type[5] = VillageType::DESERT;

        // Map piece names to internal enum
        name_to_piece["plains_fountain_01"] = VillageStartPiece::PLAINS_FOUNTAIN_01;
        name_to_piece["plains_meeting_point_1"] = VillageStartPiece::PLAINS_MEETING_POINT_1;
        name_to_piece["plains_meeting_point_2"] = VillageStartPiece::PLAINS_MEETING_POINT_2;
        name_to_piece["plains_meeting_point_3"] = VillageStartPiece::PLAINS_MEETING_POINT_3;
        name_to_piece["desert_meeting_point_1"] = VillageStartPiece::DESERT_MEETING_POINT_1;
        name_to_piece["desert_meeting_point_2"] = VillageStartPiece::DESERT_MEETING_POINT_2;
        name_to_piece["desert_meeting_point_3"] = VillageStartPiece::DESERT_MEETING_POINT_3;
        name_to_piece["savanna_meeting_point_1"] = VillageStartPiece::SAVANNA_MEETING_POINT_1;
        name_to_piece["savanna_meeting_point_2"] = VillageStartPiece::SAVANNA_MEETING_POINT_2;
        name_to_piece["savanna_meeting_point_3"] = VillageStartPiece::SAVANNA_MEETING_POINT_3;
        name_to_piece["savanna_meeting_point_4"] = VillageStartPiece::SAVANNA_MEETING_POINT_4;
        name_to_piece["taiga_meeting_point_1"] = VillageStartPiece::TAIGA_MEETING_POINT_1;
        name_to_piece["taiga_meeting_point_2"] = VillageStartPiece::TAIGA_MEETING_POINT_2;
        name_to_piece["snowy_meeting_point_1"] = VillageStartPiece::SNOWY_MEETING_POINT_1;
        name_to_piece["snowy_meeting_point_2"] = VillageStartPiece::SNOWY_MEETING_POINT_2;
        name_to_piece["snowy_meeting_point_3"] = VillageStartPiece::SNOWY_MEETING_POINT_3;
    }

    std::string get_name() const override { return "Village"; }
    ConstraintType get_type() const override { return ConstraintType::VILLAGE; }
    void initialize_device_constants() const override { /* Uses __constant__ memory directly */ }
    
    // Villages don't have a simple 20-bit filter or a reversible generation algorithm
    bool has_fast_filter() const override { return false; }
    bool has_reversing_kernel() const override { return false; }

    bool try_parse_constraint(const std::vector<std::string>& parts, Constraint& c) const override {
        // Format: ChunkX, ChunkZ, ROTATION, piece_name, biome_id, [is_abandoned]
        if (parts.size() < 5 || parts.size() > 6) return false;
        
        // Check if piece name and biome ID are valid for villages
        if (name_to_piece.find(parts[3]) == name_to_piece.end()) return false;
        int biome_id = -1;
        try { biome_id = std::stoi(parts[4]); } catch(...) { return false; }
        if (biome_id_to_type.find(biome_id) == biome_id_to_type.end()) return false;
        
        c.type = get_type();
        c.village.piece = name_to_piece.at(parts[3]);
        c.village.type = biome_id_to_type.at(biome_id);

        // Handle optional 'abandoned' flag
        c.village.is_abandoned = false;
        if (parts.size() == 6) {
            std::string abandoned_str = parts[5];
            std::transform(abandoned_str.begin(), abandoned_str.end(), abandoned_str.begin(), ::tolower);
            if (abandoned_str == "yes") {
                c.village.is_abandoned = true;
            } else if (abandoned_str != "no") {
                return false; // Invalid value for abandoned flag
            }
        }
        
        return true;
    }
};

// Concrete implementation for Shipwrecks.
class ShipwreckStructure : public IStructure {
private:
    std::map<std::string, ShipwreckType> name_to_type;
public:
    ShipwreckStructure() {
        name_to_type["rightsideup_backhalf"] = ShipwreckType::RIGHTSIDEUP_BACKHALF; name_to_type["rightsideup_backhalf_degraded"] = ShipwreckType::RIGHTSIDEUP_BACKHALF_DEGRADED;
        name_to_type["rightsideup_fronthalf"] = ShipwreckType::RIGHTSIDEUP_FRONTHALF; name_to_type["rightsideup_fronthalf_degraded"] = ShipwreckType::RIGHTSIDEUP_FRONTHALF_DEGRADED;
        name_to_type["rightsideup_full"] = ShipwreckType::RIGHTSIDEUP_FULL; name_to_type["rightsideup_full_degraded"] = ShipwreckType::RIGHTSIDEUP_FULL_DEGRADED;
        name_to_type["sideways_backhalf"] = ShipwreckType::SIDEWAYS_BACKHALF; name_to_type["sideways_backhalf_degraded"] = ShipwreckType::SIDEWAYS_BACKHALF_DEGRADED;
        name_to_type["sideways_fronthalf"] = ShipwreckType::SIDEWAYS_FRONTHALF; name_to_type["sideways_fronthalf_degraded"] = ShipwreckType::SIDEWAYS_FRONTHALF_DEGRADED;
        name_to_type["sideways_full"] = ShipwreckType::SIDEWAYS_FULL; name_to_type["sideways_full_degraded"] = ShipwreckType::SIDEWAYS_FULL_DEGRADED;
        name_to_type["upsidedown_backhalf"] = ShipwreckType::UPSIDEDOWN_BACKHALF; name_to_type["upsidedown_backhalf_degraded"] = ShipwreckType::UPSIDEDOWN_BACKHALF_DEGRADED;
        name_to_type["upsidedown_fronthalf"] = ShipwreckType::UPSIDEDOWN_FRONTHALF; name_to_type["upsidedown_fronthalf_degraded"] = ShipwreckType::UPSIDEDOWN_FRONTHALF_DEGRADED;
        name_to_type["upsidedown_full"] = ShipwreckType::UPSIDEDOWN_FULL; name_to_type["upsidedown_full_degraded"] = ShipwreckType::UPSIDEDOWN_FULL_DEGRADED;
        name_to_type["with_mast"] = ShipwreckType::WITH_MAST; name_to_type["with_mast_degraded"] = ShipwreckType::WITH_MAST_DEGRADED;
    }
    std::string get_name() const override { return "Shipwreck"; }
    ConstraintType get_type() const override { return ConstraintType::SHIPWRECK; }
    bool has_fast_filter() const override { return true; }
    bool has_reversing_kernel() const override { return true; }

    void initialize_device_constants() const override {
        std::vector<ShipwreckType> ocean_types = {
            name_to_type.at("with_mast"), name_to_type.at("upsidedown_full"), name_to_type.at("upsidedown_fronthalf"), name_to_type.at("upsidedown_backhalf"),
            name_to_type.at("sideways_full"), name_to_type.at("sideways_fronthalf"), name_to_type.at("sideways_backhalf"), name_to_type.at("rightsideup_full"),
            name_to_type.at("rightsideup_fronthalf"), name_to_type.at("rightsideup_backhalf"), name_to_type.at("with_mast_degraded"),
            name_to_type.at("upsidedown_full_degraded"), name_to_type.at("upsidedown_fronthalf_degraded"), name_to_type.at("upsidedown_backhalf_degraded"),
            name_to_type.at("sideways_full_degraded"), name_to_type.at("sideways_fronthalf_degraded"), name_to_type.at("sideways_backhalf_degraded"),
            name_to_type.at("rightsideup_full_degraded"), name_to_type.at("rightsideup_fronthalf_degraded"), name_to_type.at("rightsideup_backhalf_degraded")
        };
        std::vector<ShipwreckType> beached_types = {
            name_to_type.at("with_mast"), name_to_type.at("sideways_full"), name_to_type.at("sideways_fronthalf"), name_to_type.at("sideways_backhalf"),
            name_to_type.at("rightsideup_full"), name_to_type.at("rightsideup_fronthalf"), name_to_type.at("rightsideup_backhalf"),
            name_to_type.at("with_mast_degraded"), name_to_type.at("rightsideup_full_degraded"),
            name_to_type.at("rightsideup_fronthalf_degraded"), name_to_type.at("rightsideup_backhalf_degraded")
        };
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_STRUCTURE_LOCATION_OCEAN), ocean_types.data(), ocean_types.size() * sizeof(ShipwreckType)));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_STRUCTURE_LOCATION_BEACHED), beached_types.data(), beached_types.size() * sizeof(ShipwreckType)));
    }

    bool try_parse_constraint(const std::vector<std::string>& parts, Constraint& c) const override {
        if (parts.size() != 5) return false;
        if (name_to_type.find(parts[3]) == name_to_type.end()) return false;

        c.type = get_type();
        c.shipwreck.type = name_to_type.at(parts[3]);
        std::string biome = parts[4];
        std::transform(biome.begin(), biome.end(), biome.begin(), [](unsigned char ch){ return std::tolower(ch); });
        if (biome == "beached") c.shipwreck.isBeached = true;
        else if (biome == "ocean") c.shipwreck.isBeached = false;
        else return false;

        return true;
    }
};

// Concrete implementation for Ruined Portals.
class RuinedPortalStructure : public IStructure {
private:
    std::map<std::string, PortalType> name_to_type;
public:
    RuinedPortalStructure() {
        name_to_type["portal_1"] = PortalType::PORTAL_1; name_to_type["portal_2"] = PortalType::PORTAL_2; name_to_type["portal_3"] = PortalType::PORTAL_3;
        name_to_type["portal_4"] = PortalType::PORTAL_4; name_to_type["portal_5"] = PortalType::PORTAL_5; name_to_type["portal_6"] = PortalType::PORTAL_6;
        name_to_type["portal_7"] = PortalType::PORTAL_7; name_to_type["portal_8"] = PortalType::PORTAL_8; name_to_type["portal_9"] = PortalType::PORTAL_9;
        name_to_type["portal_10"] = PortalType::PORTAL_10;
        name_to_type["giant_portal_1"] = PortalType::GIANT_PORTAL_1; name_to_type["giant_portal_2"] = PortalType::GIANT_PORTAL_2;
        name_to_type["giant_portal_3"] = PortalType::GIANT_PORTAL_3;
    }
    std::string get_name() const override { return "Ruined Portal"; }
    ConstraintType get_type() const override { return ConstraintType::RUINED_PORTAL; }
    bool has_reversing_kernel() const override { return true; }

    void initialize_device_constants() const override { /* No __constant__ memory for portals */ }

    bool try_parse_constraint(const std::vector<std::string>& parts, Constraint& c) const override {
        if (parts.size() != 6) return false;
        if (name_to_type.find(parts[3]) == name_to_type.end()) return false;

        c.type = get_type();
        c.portal.type = name_to_type.at(parts[3]);
        std::string mirror = parts[4];
        std::transform(mirror.begin(), mirror.end(), mirror.begin(), [](unsigned char ch){ return std::tolower(ch); });
        if (mirror == "yes") c.portal.mirror = BlockMirror::FRONT_BACK;
        else if (mirror == "no") c.portal.mirror = BlockMirror::NONE;
        else return false;

        int cat = std::stoi(parts[5]);
        if (cat < 1 || cat > 3) return false;
        c.portal.category = static_cast<BiomeCategory>(cat);

        return true;
    }
};

// Manages all known structure types and parsing logic.
class StructureRegistry {
private:
    std::vector<std::unique_ptr<IStructure>> structures;
    std::map<std::string, BlockRotation> name_to_rot;

public:
    StructureRegistry() {
        structures.push_back(std::make_unique<ShipwreckStructure>());
        structures.push_back(std::make_unique<RuinedPortalStructure>());
        structures.push_back(std::make_unique<VillageStructure>()); // NEW

        name_to_rot["NONE"] = BlockRotation::NONE; name_to_rot["CLOCKWISE_90"] = BlockRotation::CLOCKWISE_90;
        name_to_rot["CLOCKWISE_180"] = BlockRotation::CLOCKWISE_180; name_to_rot["COUNTERCLOCKWISE_90"] = BlockRotation::COUNTERCLOCKWISE_90;
    }

    void initialize_all_device_constants() const {
        for (const auto& s : structures) {
            s->initialize_device_constants();
        }
    }
    
    bool parse_line(const std::string& line, Constraint& out_constraint) const {
        std::stringstream ss(line);
        std::string token;
        std::vector<std::string> parts;
        while(std::getline(ss, token, ',')) {
            token.erase(0, token.find_first_not_of(" \t"));
            token.erase(token.find_last_not_of(" \t") + 1);
            parts.push_back(token);
        }

        if (parts.size() < 4) return false;

        try {
            Constraint c;
            c.chunkX = std::stoi(parts[0]);
            c.chunkZ = std::stoi(parts[1]);
            if (name_to_rot.find(parts[2]) == name_to_rot.end()) return false;
            BlockRotation rot = name_to_rot.at(parts[2]);

            for (const auto& s : structures) {
                if (s->try_parse_constraint(parts, c)) {
                    if (c.type == ConstraintType::RUINED_PORTAL) c.portal.rotation = rot;
                    else if (c.type == ConstraintType::SHIPWRECK) c.shipwreck.rotation = rot;
                    else if (c.type == ConstraintType::VILLAGE) c.village.rotation = rot;
                    out_constraint = c;
                    return true;
                }
            }
        } catch (const std::exception&) {
            return false;
        }
        return false;
    }
};
// =======================================================================
// 5. CUDA Kernels (Global Scope)
// Kernels must be defined in the global namespace, not inside classes.
// =======================================================================

__global__ void pillarseedSearch_kernel(
    uint32_t pillarseed, const Constraint* d_constraints, int num_constraints,
    int64_t* d_found_seeds, uint32_t* d_found_count
) {
    uint32_t lowerbits = blockIdx.x;
    if (lowerbits >= 65536) return;

    int64_t partial_state = ((int64_t)pillarseed << 16) | lowerbits;
    int64_t state1 = partial_state * PILLAR_MULT + PILLAR_ADD;
    int64_t state2 = state1 * PILLAR_MULT + PILLAR_ADD;
    uint32_t half_seed_32bit = (uint32_t)((state2 ^ XOR_MASK) & 0xFFFFFFFFLL);

    StandaloneChunkRand rand;

    for (uint32_t upper16 = threadIdx.x; upper16 < 65536; upper16 += blockDim.x) {
        int64_t candidateSeed = ((int64_t)upper16 << 32) | half_seed_32bit;
        bool valid_for_all_constraints = true;
        for (int i = 0; i < num_constraints; i++) {
            bool ok;
            // MODIFIED: Added VILLAGE case
            if (d_constraints[i].type == ConstraintType::RUINED_PORTAL) {
                ok = check_portal_full(candidateSeed, d_constraints[i], rand);
            } else if (d_constraints[i].type == ConstraintType::SHIPWRECK) {
                ok = check_shipwreck_full(candidateSeed, d_constraints[i], rand);
            } else { // Village
                ok = check_village_full(candidateSeed, d_constraints[i], rand);
            }
            if (!ok) {
                valid_for_all_constraints = false;
                break;
            }
        }
        if (valid_for_all_constraints) {
            uint32_t result_idx = atomicAdd(d_found_count, 1);
            d_found_seeds[result_idx] = candidateSeed;
        }
    }
}

__device__ bool canGenerate_shipwreck_20bit_fast_filter(uint32_t lower20bits, int32_t chunkX, int32_t chunkZ) {
    int32_t regX = floorDiv(chunkX, SHIPWRECK_SPACING);
    int32_t regZ = floorDiv(chunkZ, SHIPWRECK_SPACING);
    uint32_t regionalSeed32 = (uint32_t)(((long long)lower20bits + (long long)regX * MULT_A + (long long)regZ * MULT_B + (long long)SHIPWRECK_SALT) ^ XOR_MASK);
    regionalSeed32 = (uint32_t)((long long)regionalSeed32 * LCG_MULT + LCG_ADD);
    uint32_t xCheck = (regionalSeed32 >> 17) & 3; 
    regionalSeed32 = (uint32_t)((long long)regionalSeed32 * LCG_MULT + LCG_ADD);
    uint32_t zCheck = (regionalSeed32 >> 17) & 3;
    return xCheck == (chunkX & 3) && zCheck == (chunkZ & 3);
}

__global__ void findLower20BitSeeds_kernel(const Constraint* d_shipwreck_constraints, int num_constraints, uint32_t* d_results, uint32_t* d_result_count) {
    uint32_t lower20bits = blockIdx.x * blockDim.x + threadIdx.x;
    if (lower20bits >= (1 << 20)) return;
    for (int i = 0; i < num_constraints; ++i) {
        if (!canGenerate_shipwreck_20bit_fast_filter(lower20bits, d_shipwreck_constraints[i].chunkX, d_shipwreck_constraints[i].chunkZ)) {
            return;
        }
    }
    uint32_t index = atomicAdd(d_result_count, 1);
    d_results[index] = lower20bits;
}

__global__ void reverseAndCheck_kernel(
    const uint32_t* d_valid_lower20bits, uint32_t num_valid_lower20bits,
    const Constraint* d_anchor, const Constraint* d_validators, int num_validators,
    int64_t* d_found_seeds, uint32_t* d_found_count
) {
    uint32_t lower20_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (lower20_idx >= num_valid_lower20bits) return;

    uint32_t lower20bit_seed = d_valid_lower20bits[lower20_idx];
    StandaloneChunkRand rand;
    
    // This is the validation check that is run on any found seed candidate.
    // It's a helper lambda to avoid code duplication.
    auto validate_candidate = [&](int64_t seed) -> bool {
        for(int i=0; i<num_validators; ++i) {
            bool ok;
            // MODIFIED: Added VILLAGE case
            if(d_validators[i].type == ConstraintType::RUINED_PORTAL) {
                ok = check_portal_full(seed, d_validators[i], rand);
            } else if (d_validators[i].type == ConstraintType::SHIPWRECK) {
                ok = check_shipwreck_full(seed, d_validators[i], rand);
            } else { // Village
                ok = check_village_full(seed, d_validators[i], rand);
            }
            if(!ok) return false;
        }
        return true;
    };


    if (d_anchor->type == ConstraintType::RUINED_PORTAL) {
        int32_t regX = floorDiv(d_anchor->chunkX, PORTAL_SPACING);
        int32_t regZ = floorDiv(d_anchor->chunkZ, PORTAL_SPACING);
        int32_t gen_regionsize = PORTAL_SPACING - PORTAL_SEPARATION;
        int32_t expectedRelX = ((d_anchor->chunkX % PORTAL_SPACING) + PORTAL_SPACING) % PORTAL_SPACING;
        int32_t expectedRelZ = ((d_anchor->chunkZ % PORTAL_SPACING) + PORTAL_SPACING) % PORTAL_SPACING;
        int64_t term_x = (int64_t)regX * MULT_A;
        int64_t term_z = (int64_t)regZ * MULT_B;
        uint64_t u_initial_part = (uint64_t)lower20bit_seed + (uint64_t)term_x + (uint64_t)term_z + (uint64_t)RUINED_PORTAL_SALT;
        uint64_t u_state0 = (u_initial_part ^ (uint64_t)XOR_MASK) & MASK_48;
        uint64_t u_state1 = (u_state0 * (uint64_t)LCG_MULT + (uint64_t)LCG_ADD) & MASK_48;
        uint64_t u_state2 = (u_state1 * (uint64_t)LCG_MULT + (uint64_t)LCG_ADD) & MASK_48;
        uint32_t lower20_of_state2 = (uint32_t)(u_state2 & 0xFFFFF);
        int32_t K = (lower20_of_state2 >> 17);
        int32_t R = expectedRelZ;
        int32_t i_base = (K - R % 8 + 8) % 8;
        int32_t B_base = 25 * i_base + R;
        for (int m = 0; ; ++m) {
            int64_t bits_z_64 = (int64_t)200 * m + B_base;
            if (bits_z_64 >= (1LL << 31)) break;
            int32_t bits_z = (int32_t)bits_z_64;
            if (bits_z - (bits_z % gen_regionsize) + (gen_regionsize - 1) >= 0) {
                uint64_t state2_candidate = (((uint64_t)bits_z) << 17) | (lower20_of_state2 & 0x1FFFF);
                uint64_t state1_candidate = ((state2_candidate - (uint64_t)LCG_ADD) * (uint64_t)LCG_MULT_INV) & MASK_48;
                int32_t bits_x = (int32_t)(state1_candidate >> 17);
                if (bits_x >= 0 && bits_x % gen_regionsize == expectedRelX && (bits_x - (bits_x % gen_regionsize) + (gen_regionsize - 1) >= 0)) {
                    uint64_t state0_candidate = ((state1_candidate - (uint64_t)LCG_ADD) * (uint64_t)LCG_MULT_INV) & MASK_48;
                    uint64_t u_scrambled = state0_candidate ^ (uint64_t)XOR_MASK;
                    int64_t seed = (int64_t)((u_scrambled - (uint64_t)term_x - (uint64_t)term_z - (uint64_t)RUINED_PORTAL_SALT) & MASK_48);
                    if (check_portal_full(seed, *d_anchor, rand) && validate_candidate(seed)) { 
                        d_found_seeds[atomicAdd(d_found_count, 1)] = seed; 
                    }
                }
            }
        }
    } else if (d_anchor->type == ConstraintType::SHIPWRECK) {
        int32_t regX = floorDiv(d_anchor->chunkX, SHIPWRECK_SPACING);
        int32_t regZ = floorDiv(d_anchor->chunkZ, SHIPWRECK_SPACING);
        int32_t gen_regionsize = SHIPWRECK_SPACING - SHIPWRECK_SEPARATION;
        int32_t expectedRelX = ((d_anchor->chunkX % SHIPWRECK_SPACING) + SHIPWRECK_SPACING) % SHIPWRECK_SPACING;
        int32_t expectedRelZ = ((d_anchor->chunkZ % SHIPWRECK_SPACING) + SHIPWRECK_SPACING) % SHIPWRECK_SPACING;
        int64_t term_x = (int64_t)regX * MULT_A;
        int64_t term_z = (int64_t)regZ * MULT_B;
        uint64_t u_initial_part = (uint64_t)lower20bit_seed + (uint64_t)term_x + (uint64_t)term_z + (uint64_t)SHIPWRECK_SALT;
        uint64_t u_state0 = (u_initial_part ^ (uint64_t)XOR_MASK) & MASK_48;
        uint64_t u_state1 = (u_state0 * (uint64_t)LCG_MULT + (uint64_t)LCG_ADD) & MASK_48;
        uint64_t u_state2 = (u_state1 * (uint64_t)LCG_MULT + (uint64_t)LCG_ADD) & MASK_48;
        uint32_t finalLower20LCG = (uint32_t)(u_state2 & 0xFFFFF);
        uint32_t base_Z_contrib = finalLower20LCG >> 17;
        for (uint32_t test_u = 0; test_u < 5; test_u++) {
            if ((((test_u << 3) + base_Z_contrib) % gen_regionsize) == expectedRelZ) {
                for (long j = 0; ; j++) {
                    uint64_t upper28LCG = 5 * j + test_u;
                    if (upper28LCG >= (1ULL << 28)) break;
                    uint64_t u_lcgStateForZ = (upper28LCG << 20) | finalLower20LCG;
                    uint64_t u_lcgStateForX = ((u_lcgStateForZ - (uint64_t)LCG_ADD) * (uint64_t)LCG_MULT_INV) & MASK_48;
                    if (((u_lcgStateForX >> 17) % gen_regionsize) == expectedRelX) {
                        uint64_t u_lcgInitial = ((u_lcgStateForX - (uint64_t)LCG_ADD) * (uint64_t)LCG_MULT_INV) & MASK_48;
                        uint64_t u_scrambled = u_lcgInitial ^ (uint64_t)XOR_MASK;
                        int64_t seed = (int64_t)((u_scrambled - (uint64_t)term_x - (uint64_t)term_z - (uint64_t)SHIPWRECK_SALT) & MASK_48);
                        if (check_shipwreck_full(seed, *d_anchor, rand) && validate_candidate(seed)) {
                            d_found_seeds[atomicAdd(d_found_count, 1)] = seed; 
                        }
                    }
                }
            }
        }
    }
}

__global__ void bruteforceStructureSeeds_kernel(
    const uint32_t* d_valid_lower20bits, uint32_t num_valid_lower20bits,
    const Constraint* d_constraints, int num_constraints,
    int64_t* d_found_seeds, uint32_t* d_found_count
) {
    uint64_t num_upper_bits_to_check = 1ULL << 28;
    uint64_t total_tasks = (uint64_t)num_valid_lower20bits * num_upper_bits_to_check;
    uint64_t thread_id = (uint64_t)blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = (uint64_t)gridDim.x * blockDim.x;

    StandaloneChunkRand rand;

    for (uint64_t global_idx = thread_id; global_idx < total_tasks; global_idx += stride) {
        uint32_t lower20_idx = global_idx / num_upper_bits_to_check;
        uint32_t upper28_bits = global_idx % num_upper_bits_to_check;
        
        uint32_t lower20_val = d_valid_lower20bits[lower20_idx];
        int64_t candidateSeed = ((int64_t)upper28_bits << 20) | lower20_val;
        
        bool valid_for_all = true;
        for (int i = 0; i < num_constraints; i++) {
            bool ok;
            // MODIFIED: Added VILLAGE case
            if (d_constraints[i].type == ConstraintType::RUINED_PORTAL) {
                ok = check_portal_full(candidateSeed, d_constraints[i], rand);
            } else if (d_constraints[i].type == ConstraintType::SHIPWRECK) {
                ok = check_shipwreck_full(candidateSeed, d_constraints[i], rand);
            } else { // Village
                ok = check_village_full(candidateSeed, d_constraints[i], rand);
            }
            if (!ok) {
                valid_for_all = false;
                break;
            }
        }
        if (valid_for_all) {
            uint32_t result_idx = atomicAdd(d_found_count, 1);
            d_found_seeds[result_idx] = candidateSeed;
        }
    }
}

// =======================================================================
// 6. Search Strategy Classes
// =======================================================================

class ISearchStrategy {
public:
    virtual ~ISearchStrategy() = default;
    virtual void execute(CudaBuffer<int64_t>& results_buffer, CudaBuffer<uint32_t>& count_buffer) = 0;
    virtual std::string get_description() const = 0;
};

class PillarseedSearch : public ISearchStrategy {
private:
    uint32_t pillarseed;
    CudaBuffer<Constraint> d_constraints;

public:
    PillarseedSearch(uint32_t seed, const std::vector<Constraint>& constraints) : pillarseed(seed) {
        if (constraints.empty()) {
            throw std::runtime_error("Pillarseed mode requires at least one constraint.");
        }
        d_constraints = CudaBuffer<Constraint>(constraints.size());
        d_constraints.copy_to_device(constraints.data(), constraints.size());
    }
    
    std::string get_description() const override {
        std::stringstream ss;
        ss << "--- Pillarseed Mode Activated ---\n"
           << "Using Pillarseed: " << pillarseed << "\n"
           << "With " << d_constraints.size() << " constraint(s).\n\n"
           << "Launching kernel to check 2^32 full seeds...";
        return ss.str();
    }

    void execute(CudaBuffer<int64_t>& results, CudaBuffer<uint32_t>& count) override {
        int threads_per_block = 256;
        int blocks = 65536;
        pillarseedSearch_kernel<<<blocks, threads_per_block>>>(
            pillarseed, d_constraints.get(), d_constraints.size(), results.get(), count.get()
        );
    }
};

class StandardSearch : public ISearchStrategy {
private:
    std::vector<Constraint> all_constraints;
    CudaBuffer<uint32_t> d_valid_lower20bits;
    uint32_t lower20_count = 0;

public:
    StandardSearch(const std::vector<Constraint>& constraints) : all_constraints(constraints) {}
    
    std::string get_description() const override { return "--- Standard Structure Seed Search ---"; }
    
    void run_stage1() {
        std::cout << "\n--- Stage 1: Filtering Lower 20-bit seed patterns ---\n";
        std::vector<Constraint> shipwreck_constraints;
        bool has_fast_filterable_constraint = false;
        for(const auto& c : all_constraints) {
            if(c.type == ConstraintType::SHIPWRECK) {
                shipwreck_constraints.push_back(c);
                has_fast_filterable_constraint = true;
            }
        }

        if (has_fast_filterable_constraint) {
            std::cout << "Using " << shipwreck_constraints.size() << " shipwreck(s) to filter 20-bit candidates...\n";
            CudaBuffer<Constraint> d_shipwreck_constraints(shipwreck_constraints.size());
            d_shipwreck_constraints.copy_to_device(shipwreck_constraints.data(), shipwreck_constraints.size());
            
            CudaBuffer<uint32_t> d_lower20_results(1 << 20);
            CudaBuffer<uint32_t> d_lower20_count(1);
            d_lower20_count.memset(0);
            
            int threads = 256;
            int blocks = ((1 << 20) + threads - 1) / threads;
            findLower20BitSeeds_kernel<<<blocks, threads>>>(d_shipwreck_constraints.get(), shipwreck_constraints.size(), d_lower20_results.get(), d_lower20_count.get());
            
            d_lower20_count.copy_to_host(&lower20_count, 1);
            if (lower20_count > 0) {
                d_valid_lower20bits = CudaBuffer<uint32_t>(lower20_count);
                CUDA_CHECK(hipMemcpy(d_valid_lower20bits.get(), d_lower20_results.get(), lower20_count * sizeof(uint32_t), hipMemcpyDeviceToDevice));
            }
        } else {
            std::cout << "No fast-filterable constraints (e.g., Shipwreck) provided. Generating all 2^20 candidates.\n";
            lower20_count = 1 << 20;
            std::vector<uint32_t> h_all_bits(lower20_count);
            std::iota(h_all_bits.begin(), h_all_bits.end(), 0);
            d_valid_lower20bits = CudaBuffer<uint32_t>(lower20_count);
            d_valid_lower20bits.copy_to_device(h_all_bits.data(), lower20_count);
        }
        
        std::cout << "Found " << lower20_count << " potential 20-bit candidates.\n";
    }
    
    void run_stage2_reversing(CudaBuffer<int64_t>& results, CudaBuffer<uint32_t>& count) {
        std::cout << "\n--- Stage 2: Using REVERSING Approach (1-2 constraints with reversible anchor) ---\n";
        int anchor_idx = -1;
        // Prioritize portal as anchor, then shipwreck
        for (int i = 0; i < all_constraints.size(); ++i) {
            if (all_constraints[i].type == ConstraintType::RUINED_PORTAL) { anchor_idx = i; break; }
        }
        if (anchor_idx == -1) {
            for (int i = 0; i < all_constraints.size(); ++i) {
                if (all_constraints[i].type == ConstraintType::SHIPWRECK) { anchor_idx = i; break; }
            }
        }

        Constraint h_anchor = all_constraints[anchor_idx];
        std::vector<Constraint> h_validators;
        for (int i = 0; i < all_constraints.size(); ++i) {
            if (i != anchor_idx) h_validators.push_back(all_constraints[i]);
        }
        
        CudaBuffer<Constraint> d_anchor(1);
        d_anchor.copy_to_device(&h_anchor, 1);
        CudaBuffer<Constraint> d_validators(h_validators.size());
        if (!h_validators.empty()) {
            d_validators.copy_to_device(h_validators.data(), h_validators.size());
        }

        const char* anchor_type_str = h_anchor.type == ConstraintType::RUINED_PORTAL ? "Portal" : "Shipwreck";
        std::cout << "Using " << anchor_type_str << " at [" << h_anchor.chunkX << "," << h_anchor.chunkZ << "] as anchor.\n";
        
        int threads = 256;
        int blocks = (lower20_count + threads - 1) / threads;
        reverseAndCheck_kernel<<<blocks, threads>>>(
            d_valid_lower20bits.get(), lower20_count, d_anchor.get(), d_validators.get(), 
            h_validators.size(), results.get(), count.get()
        );
    }
    
    void run_stage2_bruteforce(CudaBuffer<int64_t>& results, CudaBuffer<uint32_t>& count) {
        std::cout << "\n--- Stage 2: Using BRUTE-FORCE Approach (" << all_constraints.size() << " constraints) ---\n";
        CudaBuffer<Constraint> d_all_constraints(all_constraints.size());
        d_all_constraints.copy_to_device(all_constraints.data(), all_constraints.size());
        
        int threads = 256;
        int blocks = 32768;
        uint64_t total_tasks = (uint64_t)lower20_count * (1ULL << 28);
        std::cout << "Launching bruteforce kernel to check " << total_tasks << " total seed candidates...\n";

        bruteforceStructureSeeds_kernel<<<blocks, threads>>>(
            d_valid_lower20bits.get(), lower20_count, d_all_constraints.get(), 
            all_constraints.size(), results.get(), count.get()
        );
    }

    void execute(CudaBuffer<int64_t>& results, CudaBuffer<uint32_t>& count) override {
        run_stage1();
        if (lower20_count == 0) {
            std::cout << "No seed candidates found in Stage 1. Exiting.\n";
            return;
        }

        bool has_reversible_anchor = false;
        for (const auto& c : all_constraints) {
            if (c.type == ConstraintType::RUINED_PORTAL || c.type == ConstraintType::SHIPWRECK) {
                has_reversible_anchor = true;
                break;
            }
        }
        
        bool use_reversing = has_reversible_anchor && all_constraints.size() >= 1 && all_constraints.size() <= 10;
        if (use_reversing) {
            run_stage2_reversing(results, count);
        } else {
            run_stage2_bruteforce(results, count);
        }
    }
};

// =======================================================================
// 7. Main Application Logic
// =======================================================================

void print_usage(const char* prog_name) {
    std::cerr << "Usage: " << prog_name << " <constraints_file.txt>\n\n";
    std::cerr << "File Format (one constraint per line, '#' for comments):\n";
    std::cerr << "  - Structure-specific formats are detected automatically.\n";
    std::cerr << "  - Common Format: ChunkX, ChunkZ, ROTATION, ...specifics...\n\n";
    std::cerr << "Known Formats:\n";
    std::cerr << "  Shipwreck: ChunkX, ChunkZ, ROTATION, type_name, Ocean|Beached\n";
    std::cerr << "  Portal:    ChunkX, ChunkZ, ROTATION, portal_type, yes|no, biome_category(1-3)\n";
    std::cerr << "  Village:   ChunkX, ChunkZ, ROTATION, piece_name, biome_id, [yes|no]\n";
    std::cerr << "             -> biome_id: 1=Plains, 2=Snowy, 3=Taiga, 4=Savanna, 5=Desert\n";
    std::cerr << "             -> [yes|no] for is_abandoned is optional, defaults to 'no'.\n\n";
    std::cerr << "Pillarseed Mode:\n";
    std::cerr << "  Add the 32-bit pillarseed as the final number on its own line in the file.\n\n";
    std::cerr << "Example (Standard): -54, -14, COUNTERCLOCKWISE_90, sideways_fronthalf, Ocean\n";
    std::cerr << "Example (Village):  20, -112, NONE, taiga_meeting_point_1, 3, no\n";
}

bool load_constraints_from_file(const std::string& filename, const StructureRegistry& registry,
                                std::vector<Constraint>& constraints, int64_t& out_pillarseed) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: Could not open file '" << filename << "'" << std::endl;
        return false;
    }

    std::vector<std::string> lines;
    std::string line;
    while (std::getline(file, line)) {
        line.erase(0, line.find_first_not_of(" \t\n\r"));
        line.erase(line.find_last_not_of(" \t\n\r") + 1);
        if (line.empty() || line[0] == '#') continue;
        lines.push_back(line);
    }
    
    out_pillarseed = -1;

    if (!lines.empty()) {
        const std::string& last_line = lines.back();
        if (last_line.find(',') == std::string::npos) {
            try {
                size_t chars_processed;
                long long potential_seed = std::stoll(last_line, &chars_processed);
                if (chars_processed == last_line.length() && potential_seed >= 0 && potential_seed <= 0xFFFFFFFFLL) {
                    out_pillarseed = potential_seed;
                    lines.pop_back();
                }
            } catch (...) { /* Not a number */ }
        }
    }

    int line_num = 0;
    for (const auto& l : lines) {
        line_num++;
        Constraint c;
        if (registry.parse_line(l, c)) {
            constraints.push_back(c);
        } else {
            std::cerr << "Warning: Malformed or unknown constraint on line " << line_num << ": \"" << l << "\". Skipping." << std::endl;
        }
    }
    return !constraints.empty() || (out_pillarseed != -1);
}

int main(int argc, char** argv) {
    if (argc < 2) {
        print_usage(argv[0]);
        return 1;
    }

    auto startTime = std::chrono::high_resolution_clock::now();
    
    StructureRegistry registry;
    registry.initialize_all_device_constants();
    
    int64_t h_pillarseed = -1;
    std::vector<Constraint> h_all_constraints;
    if (!load_constraints_from_file(argv[1], registry, h_all_constraints, h_pillarseed)) {
        std::cerr << "No valid constraints or pillarseed found in file. Exiting." << std::endl;
        return 1;
    }

    const uint32_t results_buffer_size = 20000000;
    CudaBuffer<int64_t> d_found_seeds(results_buffer_size);
    CudaBuffer<uint32_t> d_found_count(1);
    d_found_count.memset(0);

    std::unique_ptr<ISearchStrategy> strategy;
    try {
        if (h_pillarseed != -1) {
            strategy = std::make_unique<PillarseedSearch>(h_pillarseed, h_all_constraints);
        } else {
            strategy = std::make_unique<StandardSearch>(h_all_constraints);
        }
    } catch (const std::runtime_error& e) {
        std::cerr << "Error initializing search strategy: " << e.what() << std::endl;
        return 1;
    }
    
    std::cout << strategy->get_description() << std::endl;
    strategy->execute(d_found_seeds, d_found_count);
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    uint32_t h_found_count = 0;
    d_found_count.copy_to_host(&h_found_count, 1);
    
    auto endTime = std::chrono::high_resolution_clock::now();
    std::cout << "\n--- Search Complete in " << std::chrono::duration<double>(endTime - startTime).count() << " seconds ---\n";

    if (h_found_count == 0) {
        std::cout << "No structure seeds found.\n";
    } else {
        if (h_found_count > results_buffer_size) {
             std::cerr << "\nFATAL ERROR: Found " << h_found_count << " seeds, which exceeds buffer size of " 
                       << results_buffer_size << ". Results are incomplete." << std::endl;
             h_found_count = results_buffer_size;
        }
        std::cout << "Found " << h_found_count << " valid seed(s). Writing to found_seeds.txt...\n";
        std::vector<int64_t> h_found_seeds(h_found_count);
        d_found_seeds.copy_to_host(h_found_seeds.data(), h_found_count);
        
        std::sort(h_found_seeds.begin(), h_found_seeds.end());
        std::ofstream outfile("found_seeds.txt");
        for (const auto& seed : h_found_seeds) outfile << seed << "\n";
        outfile.close();
        std::cout << "Done.\n";
    }
    
    return 0;
}